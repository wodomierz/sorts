#include "hip/hip_runtime.h"
#include <cstdio>
#include "../utils/cuda_device.h"
#include "../prefsum/prefsum.cuh"
extern "C" {
__global__
void prefixSum(int* in , int* out, int* prefixSums, int size, int mask) {
  __shared__ int shared[2][2048];
	int thid = 2*((blockIdx.x *  blockDim.x) + threadIdx.x);

	if (thid >= size) {
		shared[0][2*threadIdx.x] = 0;
	}
	else {
		shared[0][2*threadIdx.x] = (in[thid] >> mask)&1;
	}

	if (thid + 1>= size) {
		shared[0][2*threadIdx.x + 1] = 0;
	}
	else {
		shared[0][2*threadIdx.x + 1] = (in[thid+1] >> mask)&1;
	}

	__syncthreads();

	bool to = 0;
	prefixSumDev<1024, 2>(shared, to);

	if (thid < size) out[thid] = shared[to][2*threadIdx.x];
	if (thid +1 < size) out[thid+1] = shared[to][2*threadIdx.x +1];

	if (2*threadIdx.x +1 == (2048 - 1)) {
		prefixSums[blockIdx.x + 1] = shared[to][2*threadIdx.x +1];
	}
}


__global__
void sort(int* source, int* destination, int* prefixSum, int* prefixSums, int mask, int n,  int number_of_zeros) {
	int thid =2*(blockIdx.x *  blockDim.x) + threadIdx.x;
		if (thid < n) {
			int place;
			if ((source[thid]>>mask)&1) {
				place = number_of_zeros + prefixSum[thid] + prefixSums[blockIdx.x] - 1;
			}
			else {
				place = thid - prefixSum[thid] - prefixSums[blockIdx.x];
			}
			destination[place] = source[thid];
		}
		thid += 1024;
		if(thid < n) {
			int place;
			if ((source[thid]>>mask)&1) {
				place = number_of_zeros + prefixSum[thid] + prefixSums[blockIdx.x] - 1;
			}
			else {
				place = thid - prefixSum[thid] - prefixSums[blockIdx.x];
			}
			destination[place] = source[thid];
		}
	
	
}
}




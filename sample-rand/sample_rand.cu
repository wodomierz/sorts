#include "hip/hip_runtime.h"
#include "sample_rand.h"
#include "../prefsum/prefsum.cuh"


#include "../utils/cuda_device.h"
#include "../utils/kernel_commons.cuh"
#include "sample_rand.cuh"


extern "C" {


__device__ __forceinline__
int place(int size, int seed, int plus, int i) {
    return (i * seed + plus) % size;
}


__global__
void chujowy_sort(int *to_sort, int size) {
    chujowy_sort_dev(to_sort, size);
}


__global__
void sample(int *tab, int size, int seed, int plus, int *bst) {
    __shared__ int to_sort[SAMPLE_BLOCK];
    int thid = threadIdx.x;

    int i = thid;
    for (;i < SAMPLE_BLOCK; i+= SAMPLE_THREADS) {
        to_sort[i] = tab[place(size, seed, plus, i)];
    }

    __syncthreads();

    chujowy_sort_dev(to_sort, SAMPLE_BLOCK);
    __syncthreads();
    if (threadIdx.x ==0) {
        int iteratr = 0;
        for (int i = 2; i <= S_SIZE; i *= 2) {
            for (int j = 1; j < i; j += 2) {
                bst[iteratr++] = to_sort[(j * (S_SIZE / i) - 1)*AS];
            }
        }
        bst[S_SIZE - 1] = 0;
    }
}


__device__ __forceinline__
int findIndex(int e, int *bst) {
    int j = 1;
    int k = S_POW;
    while (k--) {
        j = 2 * j + (e > bst[j - 1]);
    }
    j = j - S_SIZE; // bucket index
    return j;
}


__global__
void prefsum1(int *localPrefsums, int *maxPrefSums, int number_of_counter_blocks, int *sample_offsets, int size) {
    prefsum1_dev<PREFSUM_THREADS, 2>(localPrefsums, maxPrefSums, number_of_counter_blocks, sample_offsets, size);
}

__global__
void prefsum(int *localPrefsums, int *maxPrefSums, int size) {
    prefsum_dev<PREFSUM_THREADS, 2>(localPrefsums, maxPrefSums, size);
}

__global__
void counters(int *to_sort, int *sample, int *prefsums, int number_of_blocks, int size) {
    __shared__ int bst[S_SIZE];
    __shared__ int histogram[A][S_SIZE];


    int x = blockIdx.x * blockDim.x * ELEMENTS_PER_THREAD + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;

    int gthid = x + y * gridDim.x * blockDim.x * ELEMENTS_PER_THREAD;

    int threadId = threadIdx.x;

    if (threadId < S_SIZE) { //?
        bst[threadId] = sample[threadId];
        for (int a = 0; a < A; ++a) {
            histogram[a][threadId] = 0;
        }
    }
    __syncthreads();


    for (int i = 0; i < ELEMENTS_PER_THREAD && gthid + i * THREADS_PER_BLOCK < size; ++i) {
//    for (int i = 0; i < ELEMENTS_PER_THREAD; ++i) {
        //ke?

        int j = findIndex(to_sort[gthid + i * THREADS_PER_BLOCK], bst);
        atomicAdd(histogram[gthid % A] + j, 1);
    }
    __syncthreads();


    if (threadId < S_SIZE) {
        //bug?
        for (int i = 1; i < A; ++i) {
            histogram[0][threadId] += histogram[i][threadId];
        }
        int index = (threadId * number_of_blocks) + blockId;
        atomicExch(prefsums + index, histogram[0][threadId]);
    }
}

__global__
void scatter(int *in, int *out, int *sample, int *prefsums, int number_of_blocks, int size) {
    __shared__ int bst[S_SIZE];
    __shared__ int histogram[S_SIZE];

    int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int y = blockIdx.y + threadIdx.y;

    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int gthid = x + y * gridDim.x * BLOCK_SIZE;
    int threadId = threadIdx.x;
    if (threadId < S_SIZE) { //?
        bst[threadId] = sample[threadId];
        histogram[threadId] = 0;
    }
    __syncthreads();

    for (int i = 0; i < ELEMENTS_PER_THREAD && gthid + i * THREADS_PER_BLOCK < size; ++i) {
        //ke?
        int e = in[gthid + i * THREADS_PER_BLOCK];
        int j = findIndex(e, bst);
        int local_index = atomicAdd(histogram + j, 1);
        int indexInPrefsums = (j * number_of_blocks) + blockId;
        int offset = 0;
        if (indexInPrefsums > 0) {
            offset = prefsums[indexInPrefsums - 1];
        }
        out[offset + local_index] = e;
    }
}

}